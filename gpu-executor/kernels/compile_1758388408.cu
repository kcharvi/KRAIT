// COMPILATION REQUEST
// Hardware: NVIDIA H100
// Backend: CUDA
// Timestamp: 1758388408
// Type: compile_only

#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

// Kernel function for matrix multiplication
__global__ void matrixMultiplyKernel(const float *A, const float *B, float *C, int widthA, int widthB) {
    // Thread index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within the matrix dimensions
    if (row < widthA && col < widthB) {
        float sum = 0.0f;
        for (int k = 0; k < widthA; ++k) {
            sum += A[row * widthA + k] * B[k * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}


int main() {
    // Matrix dimensions
    int widthA = 1024;
    int widthB = 1024;

    // Allocate host memory
    float *h_A, *h_B, *h_C;
    hipHostMalloc((void **)&h_A, widthA * widthA * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_B, widthA * widthB * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_C, widthA * widthB * sizeof(float), hipHostMallocDefault);

    // Initialize host matrices (example)
    for (int i = 0; i < widthA * widthA; ++i) h_A[i] = i % 10;
    for (int i = 0; i < widthA * widthB; ++i) h_B[i] = i % 10;


    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, widthA * widthA * sizeof(float));
    hipMalloc((void **)&d_B, widthA * widthB * sizeof(float));
    hipMalloc((void **)&d_C, widthA * widthB * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, widthA * widthA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((widthB + BLOCK_SIZE - 1) / BLOCK_SIZE, (widthA + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, widthA, widthB);

    // Copy results from device to host
    hipMemcpy(h_C, d_C, widthA * widthB * sizeof(float), hipMemcpyDeviceToHost);

    //Print a small section to verify (Avoid printing the entire matrix)
    printf("Result Matrix (first 5x5 elements):\n");
    for(int i=0; i<5; ++i){
        for(int j=0; j<5; ++j){
            printf("%f ", h_C[i*widthB + j]);
        }
        printf("\n");
    }


    //Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}