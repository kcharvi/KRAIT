// COMPILATION REQUEST
    // Hardware: NVIDIA T4
    // Backend: CUDA
    // Timestamp: 1758510530
    // Type: compile_only

    #include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

//Optimized Kernel for Matrix Multiplication
__global__ void matrixMultiplyKernel(const float *A, const float *B, float *C, int widthA, int widthB, int heightA) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < heightA && col < widthB) {
        float sum = 0.0f;
        for (int k = 0; k < widthA; ++k) {
            sum += A[row * widthA + k] * B[k * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}


int main() {
    // Matrix dimensions
    int heightA = 1024;
    int widthA = 1024;
    int widthB = 1024;

    // Allocate host memory
    float *h_A, *h_B, *h_C;
    hipHostMalloc((void **)&h_A, heightA * widthA * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_B, widthA * widthB * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_C, heightA * widthB * sizeof(float), hipHostMallocDefault);


    // Initialize host matrices (Example: fill with random values)
    for (int i = 0; i < heightA * widthA; ++i) h_A[i] = (float)rand() / RAND_MAX;
    for (int i = 0; i < widthA * widthB; ++i) h_B[i] = (float)rand() / RAND_MAX;


    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, heightA * widthA * sizeof(float));
    hipMalloc((void **)&d_B, widthA * widthB * sizeof(float));
    hipMalloc((void **)&d_C, heightA * widthB * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, heightA * widthA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);


    // Launch kernel
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((widthB + BLOCK_SIZE - 1) / BLOCK_SIZE, (heightA + BLOCK_SIZE - 1) / BLOCK_SIZE);
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, widthA, widthB, heightA);


    // Copy results from device to host
    hipMemcpy(h_C, d_C, heightA * widthB * sizeof(float), hipMemcpyDeviceToHost);


    //Print a small portion of the result for verification.  Avoid printing the entire matrix for large sizes.
    printf("Result (a small portion):\n");
    for(int i=0; i<10; ++i){
        for(int j=0; j<10; ++j){
            printf("%f ", h_C[i*widthB + j]);
        }
        printf("\n");
    }


    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    //Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}