import triton
import triton.language as tl

@triton.jit
def matmul_kernel(
    A,
    B,
    C,
    M,
    N,
    K,
    stride_am,
    stride_ak,
    stride_bk,
    stride_bn,
    stride_cm,
    stride_cn,
    BLOCK_SIZE_M: tl.constexpr,
    BLOCK_SIZE_N: tl.constexpr,
    BLOCK_SIZE_K: tl.constexpr,
):
    pid_m = tl.program_id(0)
    pid_n = tl.program_id(1)
    pid_k = tl.program_id(2)

    rm = tl.arange(0, BLOCK_SIZE_M)
    rn = tl.arange(0, BLOCK_SIZE_N)
    rk = tl.arange(0, BLOCK_SIZ