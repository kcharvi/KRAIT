// EXECUTION REQUEST
    // Hardware: NVIDIA T4
    // Backend: CUDA
    // Timestamp: 1758512458
    // Type: execute

    #define BLOCK_SIZE 32
    #include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void matrixMultiplyKernel(const float *A, const float *B, float *C, int widthA, int widthB, int heightA) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < heightA && col < widthB) {
        float sum = 0.0f;
        for (int k = 0; k < widthA; ++k) {
            sum += A[row * widthA + k] * B[k * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}


int main() {
    // Matrix dimensions
    int heightA = 1024;
    int widthA = 1024;
    int widthB = 1024;

    // Allocate host memory
    float *h_A = (float *)malloc(heightA * widthA * sizeof(float));
    float *h_B = (float *)malloc(widthA * widthB * sizeof(float));
    float *h_C = (float *)malloc(heightA * widthB * sizeof(float));

    // Initialize matrices (replace with your actual initialization)
    for (int i = 0; i < heightA * widthA; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < widthA * widthB; ++i) h_B[i] = 2.0f;


    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, heightA * widthA * sizeof(float));
    hipMalloc((void **)&d_B, widthA * widthB * sizeof(float));
    hipMalloc((void **)&d_C, heightA * widthB * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, heightA * widthA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((widthB + BLOCK_SIZE - 1) / BLOCK_SIZE, (heightA + BLOCK_SIZE - 1) / BLOCK_SIZE);
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, widthA, widthB, heightA);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy results from device to host
    hipMemcpy(h_C, d_C, heightA * widthB * sizeof(float), hipMemcpyDeviceToHost);

    // Print results (for verification - comment out for large matrices)
    //printf("Result matrix:\n");
    //for (int i = 0; i < heightA; ++i) {
    //    for (int j = 0; j < widthB; ++j) {
    //        printf("%f ", h_C[i * widthB + j]);
    //    }
    //    printf("\n");
    //}


    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}