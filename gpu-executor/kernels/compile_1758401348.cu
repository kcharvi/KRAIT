// COMPILATION REQUEST
// Hardware: NVIDIA T4
// Backend: CUDA
// Timestamp: 1758401348
// Type: compile_only

#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ void matrixMultiplyKernel(const float *A, const float *B, float *C, int widthA, int widthB) {
    // Thread index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within the matrix dimensions
    if (row < widthA && col < widthB) {
        float sum = 0.0f;
        for (int k = 0; k < widthA; ++k) {
            sum += A[row * widthA + k] * B[k * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}


int main() {
    // Matrix dimensions
    int widthA = 1024;
    int widthB = 1024;

    // Allocate host memory
    float *h_A, *h_B, *h_C;
    hipHostMalloc((void **)&h_A, widthA * widthA * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_B, widthA * widthB * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_C, widthA * widthB * sizeof(float), hipHostMallocDefault);

    // Initialize host matrices (example: fill with random values)
    for (int i = 0; i < widthA * widthA; ++i) h_A[i] = (float)rand() / RAND_MAX;
    for (int i = 0; i < widthA * widthB; ++i) h_B[i] = (float)rand() / RAND_MAX;


    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, widthA * widthA * sizeof(float));
    hipMalloc((void **)&d_B, widthA * widthB * sizeof(float));
    hipMalloc((void **)&d_C, widthA * widthB * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, widthA * widthA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((widthB + BLOCK_SIZE - 1) / BLOCK_SIZE, (widthA + BLOCK_SIZE - 1) / BLOCK_SIZE);
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, widthA, widthB);

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy results from device to host
    hipMemcpy(h_C, d_C, widthA * widthB * sizeof(float), hipMemcpyDeviceToHost);

    // Print results (optional - for smaller matrices)
    // for (int i = 0; i < widthA; ++i) {
    //     for (int j = 0; j < widthB; ++j) {
    //         printf("%f ", h_C[i * widthB + j]);
    //     }
    //     printf("\n");
    // }


    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}