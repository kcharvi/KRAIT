// EXECUTION REQUEST
    // Hardware: NVIDIA T4
    // Backend: CUDA
    // Timestamp: 1758510425
    // Type: execute

    #define BLOCK_SIZE 16
    #include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void conv2d_kernel(const float *input, const float *weight, const float *bias, float *output,
                              int in_channels, int out_channels, int in_height, int in_width, int kernel_size) {
    int out_channel = blockIdx.x * blockDim.x + threadIdx.x;
    int out_height = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_channel >= out_channels || out_height >= in_height - kernel_size + 1) return;

    int out_width;
    for (out_width = 0; out_width < in_width - kernel_size + 1; ++out_width) {
        float sum = bias[out_channel];
        for (int k = 0; k < kernel_size; ++k) {
            for (int l = 0; l < kernel_size; ++l) {
                for (int c = 0; c < in_channels; ++c) {
                    int input_index = c * in_height * in_width + (out_height + k) * in_width + (out_width + l);
                    int weight_index = out_channel * in_channels * kernel_size * kernel_size + c * kernel_size * kernel_size + k * kernel_size + l;
                    sum += input[input_index] * weight[weight_index];
                }
            }
        }
        int output_index = out_channel * in_height * in_width + out_height * in_width + out_width;
        output[output_index] = sum;
    }
}


int main() {
    // Example parameters
    int in_channels = 3;
    int out_channels = 16;
    int in_height = 224;
    int in_width = 224;
    int kernel_size = 3;

    //Input, weights, bias and output sizes
    size_t input_size = in_channels * in_height * in_width * sizeof(float);
    size_t weight_size = out_channels * in_channels * kernel_size * kernel_size * sizeof(float);
    size_t bias_size = out_channels * sizeof(float);
    size_t output_size = out_channels * in_height * in_width * sizeof(float);


    // Host memory allocation
    float *h_input = (float *)malloc(input_size);
    float *h_weight = (float *)malloc(weight_size);
    float *h_bias = (float *)malloc(bias_size);
    float *h_output = (float *)malloc(output_size);

    // Initialize host memory (replace with your actual data)
    for (size_t i = 0; i < input_size / sizeof(float); ++i) h_input[i] = i;
    for (size_t i = 0; i < weight_size / sizeof(float); ++i) h_weight[i] = i;
    for (size_t i = 0; i < bias_size / sizeof(float); ++i) h_bias[i] = i;


    // Device memory allocation
    float *d_input, *d_weight, *d_bias, *d_output;
    hipMalloc((void **)&d_input, input_size);
    hipMalloc((void **)&d_weight, weight_size);
    hipMalloc((void **)&d_bias, bias_size);
    hipMalloc((void **)&d_output, output_size);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, weight_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, bias_size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((out_channels + blockDim.x - 1) / blockDim.x, (in_height + blockDim.y -1) / blockDim.y);

    // Kernel launch
    conv2d_kernel<<<gridDim, blockDim>>>(d_input, d_weight, d_bias, d_output, in_channels, out_channels, in_height, in_width, kernel_size);

    // Copy results from device to host
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    //Error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Print results (optional - for verification)
    //printf("First 10 output values: ");
    //for (int i = 0; i < 10; ++i) printf("%f ", h_output[i]);
    //printf("\n");

    // Free memory
    free(h_input);
    free(h_weight);
    free(h_bias);
    free(h_output);
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);

    return 0;
}