#include "hip/hip_runtime.h"
__global__ void matrixMultiplyKernel(const float* A, const float* B, float* C, int m, int n, int k) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < m && col < n) {
    float sum = 0.0f;
    for (int i = 0; i < k; ++i) {
      sum += A[row * k + i] * B[i * n + col];
    }
    C[row * n + col] = sum;
  }
}


//Optimized kernel leveraging shared memory
__global__ void matrixMultiplySharedKernel(const float* A, const float* B, float* C, int m, int n, int k, int TILE_WIDTH) {
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col